#include "hip/hip_runtime.h"
/***************************************************************************************
 *                                                                                      *
 *                                     Deformetrica                                     *
 *                                                                                      *
 *    Copyright Inria and the University of Utah.  All rights reserved. This file is    *
 *    distributed under the terms of the MIT License. This file is also distributed     *
 *    under the terms of the Inria Non-Commercial License Agreement.                    *
 *                                                                                      *
 *                                                                                      *
 ****************************************************************************************/

#ifndef _GpuConv1D_cu
#define _GpuConv1D_cu

#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


//#include "CauchyFunction.h"
#include "GaussFunction.h"
#include "ScalarRadialKernel.h"

#include "GpuConv1D.h"



////////////////////////////////////////////////////////////////////////////////////////////////////
// Conv1D :
////////////////////////////////////////////////////////////////////////////////////////////////////

// Thread kernel: computation of \f$ \gamma_i = \sum_j K(x_i,y_j)\beta_j for index i given by thread id.
	template < typename TYPE, int DIMPOINT, int DIMVECT, class KER  >
__global__ void GpuConv1DOnDevice(KER Ker,
		TYPE *x, TYPE *y, TYPE *beta, TYPE *gamma,
		int nx, int ny)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ char SharedData_char[];
	TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

	TYPE xi[DIMPOINT], gammai[DIMVECT];
	if(i<nx)  // we will compute gammai only if i is in the range
	{
		// load xi from device global memory
		for(int k=0; k<DIMPOINT; k++)
			xi[k] = x[i*DIMPOINT+k];
		for(int k=0; k<DIMVECT; k++)
			gammai[k] = 0.0f;
	}

	for(int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++)
	{
		int j = tile * blockDim.x + threadIdx.x;
		if(j<ny) // we load yj and betaj from device global memory only if j<ny
		{
			int inc = DIMPOINT + DIMVECT;
			for(int k=0; k<DIMPOINT; k++)
				SharedData[threadIdx.x*inc+k] = y[j*DIMPOINT+k];
			for(int k=0; k<DIMVECT; k++)
				SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
		}
		__syncthreads();

		if(i<nx) // we compute gammai only if needed
		{
			TYPE *yj, *betaj;
			yj = SharedData;
			betaj = SharedData + DIMPOINT;
			int inc = DIMPOINT + DIMVECT;
			for(int jrel = 0; jrel < blockDim.x && jrel<ny-jstart; jrel++, yj+=inc, betaj+=inc)
				Ker.Eval(gammai,xi,yj,betaj);
		}
		__syncthreads();
	}

	// Save the result in global memory.
	if(i<nx)
		for(int k=0; k<DIMVECT; k++)
			gamma[i*DIMVECT+k] = gammai[k];
}



	template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuEvalConv1D(KER Ker, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
	// Data on the device.
	TYPE* x_d;
	TYPE* y_d;
	TYPE* beta_d;
	TYPE* gamma_d;

	// Allocate arrays on device.
	hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
	hipMalloc((void**)&y_d, sizeof(TYPE)*(ny*DIMPOINT));
	hipMalloc((void**)&beta_d, sizeof(TYPE)*(ny*DIMVECT));
	hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMVECT));

	// Send data from host to device.
	hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
	hipMemcpy(y_d, y_h, sizeof(TYPE)*(ny*DIMPOINT), hipMemcpyHostToDevice);
	hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(ny*DIMVECT), hipMemcpyHostToDevice);

	// Compute on device.
	dim3 blockSize;
	blockSize.x = 192; // number of threads in each block
	dim3 gridSize;
	gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

	/*std::cout << "1D!!!!! " << "(" << gridSize.x << "," <<gridSize.y << "," << gridSize.z <<")" << std::endl;*/

	GpuConv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
		<<<gridSize,blockSize,blockSize.x*(DIMVECT+DIMPOINT)*sizeof(TYPE)>>>
		(Ker, x_d, y_d, beta_d, gamma_d, nx, ny);

	// block until the device has completed
	hipDeviceSynchronize();

	// Send data from device to host.
	hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMVECT),hipMemcpyDeviceToHost);

	// Free memory.
	hipFree(x_d);
	hipFree(y_d);
	hipFree(beta_d);
	hipFree(gamma_d);

	return 0;
}


/*
   template < typename TYPE, int DIMPOINT, int DIMVECT >
   int CauchyGpuEvalConv1D(TYPE sigma, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
   {

   return GpuEvalConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
   (ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)),
   x_h, y_h, beta_h, gamma_h, nx, ny);
   }
 */


	template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuEvalConv1D(TYPE sigma, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{

	return GpuEvalConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
		 x_h, y_h, beta_h, gamma_h, nx, ny);
}





////////////////////////////////////////////////////////////////////////////////////////////////////
// Grad1 Conv1D :
////////////////////////////////////////////////////////////////////////////////////////////////////

	template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGrad1Conv1DOnDevice(KER Ker,
		TYPE *alpha, TYPE *x, TYPE *y, TYPE *beta, TYPE *gamma,
		int nx, int ny)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ char SharedData_char[];
	TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

	TYPE xi[DIMPOINT], alphai[DIMVECT], gammai[DIMPOINT];
	if(i<nx)  // we will compute gammai only if i is in the range
	{
		// load xi and alphai from device global memory
		for(int k=0; k<DIMPOINT; k++)
			xi[k] = x[i*DIMPOINT+k];
		for(int k=0; k<DIMVECT; k++)
			alphai[k] = alpha[i*DIMVECT+k];
		for(int k=0; k<DIMPOINT; k++)
			gammai[k] = 0.0f;
	}

	for(int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++)
	{
		int j = tile * blockDim.x + threadIdx.x;
		if(j<ny) // we load yj and betaj from device global memory only if j<ny
		{
			int inc = DIMPOINT + DIMVECT;
			for(int k=0; k<DIMPOINT; k++)
				SharedData[threadIdx.x*inc+k] = y[j*DIMPOINT+k];
			for(int k=0; k<DIMVECT; k++)
				SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
		}
		__syncthreads();
		if(i<nx) // we compute gammai only if i is in the range
		{
			TYPE *yj, *betaj;
			yj = SharedData;
			betaj = SharedData + DIMPOINT;
			int inc = DIMPOINT + DIMVECT;
			for(int jrel = 0; jrel < blockDim.x && jrel<ny-jstart; jrel++, yj+=inc, betaj+=inc)
				Ker.Grad1(gammai,alphai,xi,yj,betaj);
		}
		__syncthreads();
	}

	// Save the result in global memory.
	if(i<nx)
		for(int k=0; k<DIMPOINT; k++)
			gamma[i*DIMPOINT+k] = gammai[k];
}



	template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGrad1Conv1D(KER Ker, TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{

	// Data on the device.
	TYPE* x_d;
	TYPE* y_d;
	TYPE* alpha_d;
	TYPE* gamma_d;
	TYPE* beta_d;

	// Allocate arrays on device.
	hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
	hipMalloc((void**)&y_d, sizeof(TYPE)*(ny*DIMPOINT));
	hipMalloc((void**)&alpha_d, sizeof(TYPE)*(nx*DIMVECT));
	hipMalloc((void**)&beta_d, sizeof(TYPE)*(ny*DIMVECT));
	hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMPOINT));

	// Send data from host to device.
	hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
	hipMemcpy(y_d, y_h, sizeof(TYPE)*(ny*DIMPOINT), hipMemcpyHostToDevice);
	hipMemcpy(alpha_d, alpha_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
	hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(ny*DIMVECT), hipMemcpyHostToDevice);

	// compute on device.
	dim3 blockSize;
	blockSize.x = 192; // number of threads in each block
	dim3 gridSize;
	gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

	GpuGrad1Conv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
		<<<gridSize,blockSize,blockSize.x*(DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
		(Ker, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);

	// block until the device has completed
	hipDeviceSynchronize();

	// Send data from device to host.
	hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMPOINT),hipMemcpyDeviceToHost);

	// Free memory.
	hipFree(x_d);
	hipFree(y_d);
	hipFree(alpha_d);
	hipFree(gamma_d);
	hipFree(beta_d);

	return 0;
}


/*
   template < typename TYPE, int DIMPOINT, int DIMVECT >
   int CauchyGpuGrad1Conv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
   {
   return GpuGrad1Conv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
   (ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)),
   alpha_h, x_h, y_h, beta_h, gamma_h, nx, ny);
   }
 */



	template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGrad1Conv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
	return GpuGrad1Conv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
		 alpha_h, x_h, y_h, beta_h, gamma_h, nx, ny);
}





////////////////////////////////////////////////////////////////////////////////////////////////////
// Grad Conv1D :
////////////////////////////////////////////////////////////////////////////////////////////////////

	template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGradConv1DOnDevice(KER Ker,
		TYPE *x, TYPE *beta, unsigned int dim, TYPE *gamma,
		int nx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ char SharedData_char[];
	TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

	TYPE xi[DIMPOINT], gammai[DIMVECT];
	if(i<nx)  // we will compute gammai only if i is in the range
	{
		// load xi from device global memory
		for(int k=0; k<DIMPOINT; k++)
			xi[k] = x[i*DIMPOINT+k];
		for(int k=0; k<DIMVECT; k++)
			gammai[k] = 0.0f;
	}

	for(int jstart = 0, tile = 0; jstart < nx; jstart += blockDim.x, tile++)
	{
		int j = tile * blockDim.x + threadIdx.x;
		if(j<nx) // we load xj and betaj from device global memory only if j<nx
		{
			int inc = DIMPOINT + DIMVECT;
			for(int k=0; k<DIMPOINT; k++)
				SharedData[threadIdx.x*inc+k] = x[j*DIMPOINT+k];
			for(int k=0; k<DIMVECT; k++)
				SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
		}
		__syncthreads();
		if(i<nx) // we compute gammai only if i is in the range
		{
			TYPE *xj, *betaj;
			xj = SharedData;
			betaj = SharedData + DIMPOINT;
			int inc = DIMPOINT + DIMVECT;
			for(int jrel = 0; jrel < blockDim.x && jrel<nx-jstart; jrel++, xj+=inc, betaj+=inc)
				Ker.Graddim(gammai, xi, xj,  betaj, &dim);
		}
		__syncthreads();
	}

	// Save the result in global memory.
	if(i<nx)
		for(int k=0; k<DIMVECT; k++)
			gamma[i*DIMVECT+k] = gammai[k];
}



	template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGradConv1D(KER Ker,
		TYPE* x_h, TYPE* beta_h,unsigned int dim_h, TYPE* gamma_h,
		int nx)
{

	// Data on the device.
	TYPE* x_d;
	TYPE* gamma_d;
	TYPE* beta_d;

	// Allocate arrays on device.
	hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
	hipMalloc((void**)&beta_d, sizeof(TYPE)*(nx*DIMVECT));
	hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMVECT));

	// Send data from host to device.
	hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
	hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);

	// compute on device.
	dim3 blockSize;
	blockSize.x = 192; // number of threads in each block
	dim3 gridSize;
	gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

	GpuGradConv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
		<<<gridSize,blockSize,blockSize.x*(DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
		(Ker, x_d, beta_d,dim_h, gamma_d, nx);

	// block until the device has completed
	hipDeviceSynchronize();

	// Send data from device to host.
	hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMVECT),hipMemcpyDeviceToHost);

	// Free memory.
	hipFree(x_d);
	hipFree(beta_d);
	hipFree(gamma_d);

	return 0;
}



//template < typename TYPE, int DIMPOINT, int DIMVECT >
//int CauchyGpuGradConv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* beta_h, TYPE* gamma_h, int nx)
//{
//	return GpuGradConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
//		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)),
//			alpha_h, x_h, beta_h, gamma_h, nx);
//}



	template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGradConv1D(TYPE sigma,  TYPE* x_h, TYPE* beta_h, unsigned int dim_h, TYPE* gamma_h, int nx)
{
	return GpuGradConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
		 x_h, beta_h,dim_h,  gamma_h, nx);
}






////////////////////////////////////////////////////////////////////////////////////////////////////
// Grad_varlin Conv1D :
////////////////////////////////////////////////////////////////////////////////////////////////////

	template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGradConv_varlin_1DOnDevice(KER Ker,
		TYPE *x, TYPE *y, TYPE *beta, TYPE *gamma,
		int nx, int ny)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ char SharedData_char[];
	TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

	TYPE xi[DIMPOINT], gammai[DIMVECT*DIMPOINT];
	if(i<nx)  // we will compute gammai only if i is in the range
	{
		// load xi from device global memory
		for(int k=0; k<DIMPOINT; k++)
			xi[k] = x[i*DIMPOINT+k];
		for(int k=0; k<DIMVECT*DIMPOINT; k++)
			gammai[k] = 0.0f;
	}

	for(int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++)
	{
		int j = tile * blockDim.x + threadIdx.x;
		if(j<ny) // we load yj and betaj from device global memory only if j<ny
		{
			int inc = DIMPOINT + DIMVECT;
			for(int k=0; k<DIMPOINT; k++)
				SharedData[threadIdx.x*inc+k] = y[j*DIMPOINT+k];
			for(int k=0; k<DIMVECT; k++)
				SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
		}
		__syncthreads();
		if(i<nx) // we compute gammai only if i is in the range
		{
			TYPE *yj, *betaj;
			yj = SharedData;
			betaj = SharedData + DIMPOINT;
			int inc = DIMPOINT + DIMVECT;
			for(int jrel = 0; jrel < blockDim.x && jrel<ny-jstart; jrel++, yj+=inc, betaj+=inc)
				Ker.Grad_varlin(gammai, xi, yj,  betaj);
		}
		__syncthreads();
	}

	// Save the result in global memory.
	if(i<nx)
		for(int k=0; k<DIMVECT*DIMPOINT; k++)
			gamma[i*DIMVECT*DIMPOINT+k] = gammai[k];
}



	template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGradConv_varlin_1D(KER Ker,
		TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h,
		int nx, int ny)
{

	// Data on the device.
	TYPE* x_d;
	TYPE* y_d;
	TYPE* gamma_d;
	TYPE* beta_d;

	// Allocate arrays on device.
	hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
	hipMalloc((void**)&y_d, sizeof(TYPE)*(ny*DIMPOINT));
	hipMalloc((void**)&beta_d, sizeof(TYPE)*(ny*DIMVECT));
	hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMVECT*DIMPOINT));

	// Send data from host to device.
	hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
	hipMemcpy(y_d, y_h, sizeof(TYPE)*(ny*DIMPOINT), hipMemcpyHostToDevice);
	hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(ny*DIMVECT), hipMemcpyHostToDevice);

	// compute on device.
	dim3 blockSize;
	blockSize.x = 192; // number of threads in each block
	dim3 gridSize;
	gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

	GpuGradConv_varlin_1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
		<<<gridSize,blockSize,blockSize.x*(DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
		(Ker, x_d, y_d, beta_d, gamma_d, nx, ny);

	// block until the device has completed
	hipDeviceSynchronize();

	// Send data from device to host.
	hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMVECT*DIMPOINT),hipMemcpyDeviceToHost);

	// Free memory.
	hipFree(x_d);
	hipFree(y_d);
	hipFree(beta_d);
	hipFree(gamma_d);

	return 0;
}



//template < typename TYPE, int DIMPOINT, int DIMVECT >
//int CauchyGpuGradConv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* beta_h, TYPE* gamma_h, int nx)
//{
//	return GpuGradConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
//		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)),
//			alpha_h, x_h, beta_h, gamma_h, nx);
//}



	template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGradConv_varlin_1D(TYPE sigma,  TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
	return GpuGradConv_varlin_1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
		 x_h, y_h, beta_h, gamma_h, nx, ny);
}






////////////////////////////////////////////////////////////////////////////////////////////////////
// GradDiff Conv1D :
////////////////////////////////////////////////////////////////////////////////////////////////////

	template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGradDiffConv1DOnDevice(KER Ker,
		TYPE *x, TYPE *beta, TYPE *eta, TYPE *gamma,
		int nx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ char SharedData_char[];
	TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

	TYPE xi[DIMPOINT], betai[DIMVECT], etai[DIMPOINT], gammai[DIMPOINT];
	if(i<nx)  // we will compute gammai only if i is in the range
	{
		// load xi, etai, betai from device global memory
		for(int k=0; k<DIMPOINT; k++)
			xi[k] = x[i*DIMPOINT+k];
		for(int k=0; k<DIMVECT; k++)
			betai[k] = beta[i*DIMVECT+k];
		for(int k=0; k<DIMPOINT; k++)
			etai[k] = eta[i*DIMPOINT+k];
		for(int k=0; k<DIMPOINT; k++)
			gammai[k] = 0.0f;
	}

	for(int jstart = 0, tile = 0; jstart < nx; jstart += blockDim.x, tile++)
	{
		int j = tile * blockDim.x + threadIdx.x;
		if(j<nx) // we load xj, etaj and betaj from device global memory only if j<nx
		{
			int inc = 2 * DIMPOINT + DIMVECT;
			for(int k=0; k<DIMPOINT; k++)
				SharedData[threadIdx.x*inc+k] = x[j*DIMPOINT+k];
			for(int k=0; k<DIMVECT; k++)
				SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
			for(int k=0; k<DIMPOINT; k++)
				SharedData[threadIdx.x*inc+DIMPOINT+DIMVECT+k] = eta[j*DIMPOINT+k];
		}
		__syncthreads();
		if(i<nx) // we compute gammai only if i is in the range
		{
			TYPE *xj, *betaj, *etaj;
			xj = SharedData;
			betaj = SharedData + DIMPOINT;
			etaj = SharedData + DIMPOINT + DIMVECT;
			int inc = 2 * DIMPOINT + DIMVECT;
			for(int jrel = 0; jrel < blockDim.x && jrel<nx-jstart; jrel++, xj+=inc, betaj+=inc, etaj+=inc)
				Ker.GradDiff(gammai, xi, xj, betai, betaj, etai, etaj);
		}
		__syncthreads();
	}

	// Save the result in global memory.
	if(i<nx)
		for(int k=0; k<DIMPOINT; k++)
			gamma[i*DIMPOINT+k] = gammai[k];
}



	template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGradDiffConv1D(KER Ker,
		TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h,
		int nx)
{

	// Data on the device.
	TYPE* x_d;
	TYPE* beta_d;
	TYPE* gamma_d;
	TYPE* eta_d;

	// Allocate arrays on device.
	hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
	hipMalloc((void**)&beta_d, sizeof(TYPE)*(nx*DIMVECT));
	hipMalloc((void**)&eta_d, sizeof(TYPE)*(nx*DIMPOINT));
	hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMPOINT));

	// Send data from host to device.
	hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
	hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
	hipMemcpy(eta_d, eta_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);

	// compute on device.
	dim3 blockSize;
	blockSize.x = 192; // number of threads in each block
	dim3 gridSize;
	gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

	GpuGradDiffConv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
		<<<gridSize,blockSize,blockSize.x*(2*DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
		(Ker, x_d, beta_d, eta_d, gamma_d, nx);

	// block until the device has completed
	hipDeviceSynchronize();

	// Send data from device to host.
	hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMPOINT),hipMemcpyDeviceToHost);

	// Free memory.
	hipFree(x_d);
	hipFree(eta_d);
	hipFree(beta_d);
	hipFree(gamma_d);

	return 0;
}


/*
   template < typename TYPE, int DIMPOINT, int DIMVECT >
   int CauchyGpuGradDiffConv1D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
   {
   return GpuGradDiffConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
   (ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)),
   x_h, beta_h, eta_h, gamma_h, nx);
   }
 */


	template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGradDiffConv1D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
{
	return GpuGradDiffConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
		 x_h, beta_h, eta_h, gamma_h, nx);
}



// http://www.parashift.com/c++-faq-lite/separate-template-fn-defn-from-decl.html
#define DECLARE_Conv1DS(TYPE,DIMPOINT,DIMVECT) \
	template int GaussGpuEvalConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int, int); \
template int GaussGpuGrad1Conv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, TYPE*, int, int); \
template int GaussGpuGradConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, unsigned int, TYPE*, int); \
template int GaussGpuGradConv_varlin_1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int, int); \
template int GaussGpuGradDiffConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int); 
#define DECLARE_Conv1DS_ALLDIMS_FOR(TYPE) \
	DECLARE_Conv1DS(TYPE,1,1) \
DECLARE_Conv1DS(TYPE,2,1) \
DECLARE_Conv1DS(TYPE,2,2) \
DECLARE_Conv1DS(TYPE,2,3) \
DECLARE_Conv1DS(TYPE,2,4) \
DECLARE_Conv1DS(TYPE,3,1) \
DECLARE_Conv1DS(TYPE,3,3) \
	DECLARE_Conv1DS(TYPE,3,6)
	DECLARE_Conv1DS_ALLDIMS_FOR(float)
DECLARE_Conv1DS_ALLDIMS_FOR(double)




#endif /* _GpuConv1D_cu */
